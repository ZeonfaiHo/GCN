#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <chrono>

#define DIM_M 1024
#define DIM_N 64
#define DIM_K 1024

int main() {
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    int size_A = DIM_M * DIM_K * sizeof(float);
    int size_B = DIM_K * DIM_N * sizeof(float);
    int size_C = DIM_M * DIM_N * sizeof(float);

    hipHostMalloc((void**)&h_A, size_A, hipHostMallocDefault);
    hipHostMalloc((void**)&h_B, size_B, hipHostMallocDefault);
    hipHostMalloc((void**)&h_C, size_C, hipHostMallocDefault);

    for(int i = 0; i < DIM_M * DIM_K; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
    }
    for(int i = 0; i < DIM_K * DIM_N; i++) {
        h_B[i] = rand() / (float)RAND_MAX;
    }

    auto start = std::chrono::high_resolution_clock::now();

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, DIM_N, DIM_M, DIM_K, &alpha, d_B, DIM_N, d_A, DIM_K, &beta, d_C, DIM_N);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<float, std::milli> duration_ms = end - start;
    printf("Matrix multiplication time: %f ms\n", duration_ms.count());

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
